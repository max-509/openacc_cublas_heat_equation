#include "hip/hip_runtime.h"
#include "heat_equation_solver_impl.h"

#include <stdexcept>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <algorithm>

#undef fabs
#undef fmax

#include <hipblas.h>

#include <type_traits>

#pragma acc routine seq
template <typename T>
constexpr T generic_abs(T v)
{
  if constexpr (std::is_same_v<T, float>)
  {
    return fabsf(v);
  }
  else
  {
    return fabs(v);
  }
}

#if (TARGET_DEVICE == GPU)

#define CUBLAS_CHECK(err)                                                           \
  do                                                                                \
  {                                                                                 \
    hipblasStatus_t err_ = (err);                                                    \
    if (err_ != HIPBLAS_STATUS_SUCCESS)                                              \
    {                                                                               \
      std::fprintf(stderr, "cublas error %d at %s:%d\n", err_, __FILE__, __LINE__); \
      throw std::runtime_error("cublas error");                                     \
    }                                                                               \
  } while (0)

template <typename T>
hipblasStatus_t cublasCopy(hipblasHandle_t handle,
                          int n,
                          const T *x,
                          int incx,
                          T *y,
                          int incy)
{
  if constexpr (std::is_same_v<float, T>)
  {
    return hipblasScopy(handle, n, x, incx, y, incy);
  }
  else
  {
    return hipblasDcopy(handle, n, x, incx, y, incy);
  }
}

template <typename T>
hipblasStatus_t cublasAxpy(hipblasHandle_t handle,
                          int n,
                          const T *alpha, /* host or device pointer */
                          const T *x,
                          int incx,
                          T *y,
                          int incy)
{
  if constexpr (std::is_same_v<float, T>)
  {
    return hipblasSaxpy(handle, n, alpha, x, incx, y, incy);
  }
  else
  {
    return hipblasDaxpy(handle, n, alpha, x, incx, y, incy);
  }
}

template <typename T>
hipblasStatus_t cublasIamax(hipblasHandle_t handle,
                           int n,
                           const T *x,
                           int incx,
                           int *result)
{
  if constexpr (std::is_same_v<float, T>)
  {
    return hipblasIsamax(handle, n, x, incx, result);
  }
  else
  {
    return hipblasIdamax(handle, n, x, incx, result);
  }
}

template <typename T>
struct err_computer
{
  T operator()(const T *__restrict__ buff_grid, T *__restrict__ diff_buff,
               const size_t grid_size, hipblasHandle_t handle)
  {
    const size_t grid_sqr = grid_size * grid_size;

    T err;
    T a;
    int err_idx;

#pragma acc declare create(err) device_resident(a, err_idx)
    {

#pragma acc data present(a)
#pragma acc kernels num_gangs(1) num_workers(1)
      a = static_cast<T>(-1.0);

#pragma acc host_data use_device(buff_grid, diff_buff)
      {
        CUBLAS_CHECK(cublasCopy(handle, grid_sqr, buff_grid, 1, diff_buff, 1));
#pragma acc host_data use_device(a)
        CUBLAS_CHECK(cublasAxpy(handle, grid_sqr, &a, buff_grid + grid_sqr, 1, diff_buff, 1));
      }

#pragma acc host_data use_device(err_idx, diff_buff)
      CUBLAS_CHECK(cublasIamax(handle, grid_sqr, diff_buff, 1, &err_idx));
#pragma acc data present(err, err_idx, diff_buff)
#pragma acc kernels num_gangs(1) num_workers(1)
      err = diff_buff[err_idx];
#pragma acc update host(err)
    }

    return generic_abs(err);
  }
};

#else

template <typename T>
struct err_computer
{
  T operator()(T *__restrict__ buff_grid, T *__restrict__ diff_buff /*not used*/,
               const size_t grid_size, hipblasHandle_t handle)
  {
    const size_t grid_sqr = grid_size * grid_size;
    T err = 0.0;
#pragma acc wait

#pragma acc data present(buff_grid [0:grid_sqr * 2])
    {
#pragma acc kernels
      {
#pragma acc loop independent collapse(2) reduction(max \
                                                   : err)
        for (size_t i = 1; i < grid_size - 1; ++i)
        {
          for (size_t j = 1; j < grid_size - 1; ++j)
          {
            const size_t grid_idx = i * grid_size + j;
            err = max(err, abs(buff_grid[grid_idx] - buff_grid[grid_sqr + grid_idx]));
          }
        }
      }
    }

    return err;
  }
};

#endif // TARGET_DEVICE

#ifndef N_ERR_COMPUTING_IN_DEVICE
#define N_ERR_COMPUTING_IN_DEVICE 1500
#endif // N_ERR_COMPUTING_IN_DEVICE

int solve_heat_equation(FLOAT_TYPE *__restrict__ init_grid, const size_t grid_size, const size_t max_iter, const FLOAT_TYPE etol, size_t *last_iter, FLOAT_TYPE *last_etol)
{
  const size_t grid_sqr = grid_size * grid_size;
  FLOAT_TYPE *__restrict__ buff_grid = (FLOAT_TYPE *)malloc(sizeof(FLOAT_TYPE) * (grid_sqr * 2));
  if (NULL == buff_grid)
  {
    return 1;
  }

  hipblasHandle_t cublas_handle = NULL;
  if (HIPBLAS_STATUS_SUCCESS != hipblasCreate(&cublas_handle))
  {
    free(buff_grid);
    return 1;
  }

  hipblasSetPointerMode(cublas_handle, HIPBLAS_POINTER_MODE_DEVICE);

#pragma acc data copy(init_grid [0:grid_sqr]) create(buff_grid [0:grid_sqr * 2])
  {
    FLOAT_TYPE err = (FLOAT_TYPE)INFINITY;

    size_t curr_iter;

#pragma acc data present(init_grid [0:grid_sqr], buff_grid [0:grid_sqr * 2])
#pragma acc parallel
    {
#pragma acc loop independent collapse(2)
      for (size_t i = 0; i < grid_size; ++i)
      {
        for (size_t j = 0; j < grid_size; ++j)
        {
          size_t grid_idx = i * grid_size + j;
          buff_grid[grid_idx] = init_grid[grid_idx];
          buff_grid[grid_sqr + grid_idx] = init_grid[grid_idx];
        }
      }
    }

    size_t n_err_iter;
    for (curr_iter = 0u; curr_iter < max_iter && err > etol; curr_iter += n_err_iter)
    {

      for (n_err_iter = 0; n_err_iter < N_ERR_COMPUTING_IN_DEVICE; n_err_iter += 2)
      {
#pragma acc data present(buff_grid [0:grid_sqr * 2])
#pragma acc kernels async
        {
#pragma acc loop independent collapse(2)
          for (size_t i = 1; i < grid_size - 1; ++i)
          {
            for (size_t j = 1; j < grid_size - 1; ++j)
            {
              const size_t grid_idx = i * grid_size + j;
              buff_grid[grid_sqr + grid_idx] = (FLOAT_TYPE)0.25 * (buff_grid[grid_idx - grid_size] +
                                                                   buff_grid[grid_idx + grid_size] +
                                                                   buff_grid[grid_idx - 1] +
                                                                   buff_grid[grid_idx + 1]);
            }
          }

#pragma acc loop independent collapse(2)
          for (size_t i = 1; i < grid_size - 1; ++i)
          {
            for (size_t j = 1; j < grid_size - 1; ++j)
            {
              const size_t grid_idx = i * grid_size + j;
              const size_t next_grid_idx = grid_sqr + grid_idx;
              buff_grid[grid_idx] = (FLOAT_TYPE)0.25 * (buff_grid[next_grid_idx - grid_size] +
                                                        buff_grid[next_grid_idx + grid_size] +
                                                        buff_grid[next_grid_idx - 1] +
                                                        buff_grid[next_grid_idx + 1]);
            }
          }
        }
      }

      err = err_computer<FLOAT_TYPE>{}(buff_grid, init_grid, grid_size, cublas_handle);
    }

#pragma acc wait

#pragma acc data present(init_grid [0:grid_sqr], buff_grid [0:grid_sqr * 2])
#pragma acc parallel
    {
#pragma acc loop independent collapse(2)
      for (size_t i = 0; i < grid_size; ++i)
      {
        for (size_t j = 0; j < grid_size; ++j)
        {
          size_t grid_idx = i * grid_size + j;
          init_grid[grid_idx] = buff_grid[grid_idx];
        }
      }
    }

    if (NULL != last_iter)
    {
      *last_iter = curr_iter;
    }
    if (NULL != last_etol)
    {
      *last_etol = err;
    }
  }

  hipblasDestroy(cublas_handle);
  free(buff_grid);

  return 0;
}

const char *get_solver_version()
{
  return "BLAS device pointer mode";
}
